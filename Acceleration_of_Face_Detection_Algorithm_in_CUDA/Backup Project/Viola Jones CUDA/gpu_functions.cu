#include "hip/hip_runtime.h"
#include "gpu_functions.cuh"
#include <stdio.h>

/** CUDA kernel to add two vectors*/

// USELESS
__global__  void integralImageRows_GPU(int *sumData_cuda, int *sqsumData_cuda, unsigned char *data_cuda, int width, int height){

	int bidx = blockIdx.x*THREADS2+threadIdx.x;

	int i;
	unsigned char temp;

	int sh = 0, sqh = 0;

	if (bidx < height){
		for (i = 0; i<width; i++){
			temp = data_cuda[bidx*width + i];
			sh += temp;
			sqh += temp*temp;
			sumData_cuda[bidx*width + i] = sh;
			sqsumData_cuda[bidx*width + i] = sqh;
		}
	}
}

__global__  void integralImageCols_GPU(int *sumData_cuda, int *sqsumData_cuda, int width, int height){	

	int bidx = blockIdx.x*THREADS2+threadIdx.x;

	int i;

	if (bidx < width){
		for (i = 1; i<height; i++){
			sumData_cuda[width*i + bidx] += sumData_cuda[width*(i-1) + bidx];
			sqsumData_cuda[width*i + bidx] += sqsumData_cuda[width*(i-1) + bidx];
		}
	}

}



__global__ void runCascadeClassifier(int* result_cuda, int start_stage, int cascade_n_stages, int cascade_inv_window_area, int cascade_sum_width, int cascade_sqsum_width, int x2, int y2,
									int* cascade_pq0_cuda, int cascade_pq1, int cascade_pq2, int cascade_pq3, 
									int* cascade_p0_cuda, int cascade_p1, int cascade_p2, int cascade_p3,
									int* tree_thresh_array_cuda, int* scaled_rectangles_array_cuda, int* weights_array_cuda, int* alpha1_array_cuda, int* alpha2_array_cuda, int* stages_thresh_array_cuda, int* stages_array_cuda)
{

	int p_offset, pq_offset;
	int i, j;
	unsigned int mean;
	unsigned int variance_norm_factor;
	int haar_counter = 0;
	int w_index = 0;
	int r_index = 0;
	int stage_sum;
	bool end = false;

	int* cascade_pq1_cuda = cascade_pq0_cuda + cascade_pq1;
	int* cascade_pq2_cuda = cascade_pq0_cuda + cascade_pq2;
	int* cascade_pq3_cuda = cascade_pq0_cuda + cascade_pq3;
	int* cascade_p1_cuda = cascade_p0_cuda + cascade_p1;
	int* cascade_p2_cuda = cascade_p0_cuda + cascade_p2;
	int* cascade_p3_cuda = cascade_p0_cuda + cascade_p3;

	// blockId
	int ptx = blockIdx.x*blockDim.x + threadIdx.x;
	int pty = blockIdx.y*blockDim.y + threadIdx.y;

	if (ptx <= x2 && pty <= y2){

		int index = pty*x2 + ptx;
		
		p_offset = pty * (cascade_sum_width) + ptx;
		pq_offset = pty * (cascade_sqsum_width) + ptx;

		variance_norm_factor =  (cascade_pq0_cuda[pq_offset] - cascade_pq1_cuda[pq_offset] - cascade_pq2_cuda[pq_offset] + cascade_pq3_cuda[pq_offset]);
		mean = (cascade_p0_cuda[p_offset] - cascade_p1_cuda[p_offset] - cascade_p2_cuda[p_offset] + cascade_p3_cuda[p_offset]);

		variance_norm_factor = (variance_norm_factor*cascade_inv_window_area);
		variance_norm_factor =  variance_norm_factor - mean*mean;

		if( variance_norm_factor > 0 )
			variance_norm_factor = (int)sqrtf((float)variance_norm_factor);		// GPU Function
		else
			variance_norm_factor = 1;

		for( i = start_stage; i < cascade_n_stages; i++ )
			{

				stage_sum = 0;

				for( j = 0; j < stages_array_cuda[i]; j++ )
				{

					// the node threshold is multiplied by the standard deviation of the image 
					int t = tree_thresh_array_cuda[haar_counter] * variance_norm_factor;
					int sum;


					sum = (*(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index] + p_offset)
							 - *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 1] + p_offset)
							 - *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 2] + p_offset)
							 + *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 3] + p_offset))
						* weights_array_cuda[w_index];


					sum += (*(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index+4] + p_offset)
						- *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 5] + p_offset)
						- *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 6] + p_offset)
						+ *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 7] + p_offset))
						* weights_array_cuda[w_index + 1];

					if ((scaled_rectangles_array_cuda[r_index+8] != 0))
						sum += (*(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index+8] + p_offset)
							- *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 9] + p_offset)
							- *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 10] + p_offset)
							+ *(cascade_p0_cuda + scaled_rectangles_array_cuda[r_index + 11] + p_offset))
							* weights_array_cuda[w_index + 2];

					if(sum >= t)
						stage_sum += alpha2_array_cuda[haar_counter];
					else
						stage_sum += alpha1_array_cuda[haar_counter];

					

					//n_features++;
					haar_counter++;
					w_index+=3;
					r_index+=12;
				} // end of j loop 


				if( stage_sum < 0.4*stages_thresh_array_cuda[i] ){
				 	result_cuda[index] =  -i;
				 	end = true;

				 	break;
				} // end of the per-stage thresholding 
			} // end of i loop 

		if (!end)
			result_cuda[index] = 1;

	}

}